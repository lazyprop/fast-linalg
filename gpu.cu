#include "hip/hip_runtime.h"
#include <iostream>
#include <memory>
#include <chrono>

#include "util.h"
#include "matmul.h"

#define CHECK_ERROR() \
  { \
    hipError_t e = hipGetLastError(); \
    std::cout << hipGetErrorName(e) << '\n'; \
    std::cout << hipGetErrorString(e) << '\n'; \
  }

#define BENCH_KERNEL(name, N, kernel, grid_dim, block_dim, d_a, d_b, d_c)  \
  { \
    const int runs = 100; \
    double ms = 0; \
    for (int i = 0; i < runs; i++) { \
      zero_cuda<N>(d_c); \
      hipDeviceSynchronize(); \
      auto start = std::chrono::high_resolution_clock::now(); \
      kernel<<<grid_dim, block_dim>>>(d_a, d_b, d_c);                     \
      hipDeviceSynchronize(); \
      auto end = std::chrono::high_resolution_clock::now();            \
      std::chrono::duration<double, std::milli> ms_double = end - start; \
      ms += ms_double.count();                                     \
    } \
    std::cout << name << ": " << (2.0 * N * N * N) / (1e9 * ms / runs / 1000) \
              << " GFLOPS/s\n"; \
  }


#define DEBUG_KERNEL(name, N, kernel, grid_dim, block_dim, d_a, d_b, d_c)  \
  { \
    kernel<<<grid_dim, block_dim>>>(d_a, d_b, d_c);                     \
    hipDeviceSynchronize(); \
    CHECK_ERROR(); \
  }




template<int N>
__global__ void baseline_cuda(float* a, float* b, float* c) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  int j = blockIdx.y * blockDim.y + threadIdx.y;
  float acc = 0;
  for (int k = 0; k < N; k++) {
    acc += a[i*N+k] * b[k*N+j];
  }
  c[i*N+j] = acc;
}


template<int N, int Mc>
__global__ void gmem_coalesced(float* a, float* b, float* c) {
  int i = blockIdx.x * Mc + (threadIdx.x / Mc);
  int j = blockIdx.y * Mc + (threadIdx.x % Mc);
  float acc = 0;
  for (int k = 0; k < N; k++) {
    acc += a[i*N+k] * b[k*N+j];
  }
  c[i*N+j] = acc;
}

template<int N, int Mc>
__global__ void smem_blocked(float* a, float* b, float* c) {
  int i = blockIdx.x * Mc;
  int j = blockIdx.y * Mc;
  int ii = threadIdx.x / Mc;
  int jj = threadIdx.x % Mc;

  __shared__ float aa[Mc][Mc], bb[Mc][Mc], cc[Mc][Mc];
  cc[ii][jj] = 0;
  for (int k = 0; k < N; k += Mc) {
    // abusing the index notation here
    // basically each thread loads one Mc x Mc element of a and b
    aa[ii][jj] = a[(i+ii)*N+k+jj];
    bb[ii][jj] = b[(k+ii)*N+j+jj];

    __syncthreads();
    for (int kk = 0; kk < Mc; kk++) {
      cc[ii][jj] += aa[ii][kk] * bb[kk][jj];
    }
    __syncthreads();
  }

  c[(i+ii)*N+j+jj] = cc[ii][jj];
}

template<int N, int Mc>
__global__ void smem_blocked2(float* a, float* b, float* c) {
  int i = blockIdx.x * Mc;
  int j = blockIdx.y * Mc;
  int ii = threadIdx.x / Mc;
  int jj = threadIdx.x % Mc;

  __shared__ float aa[Mc][Mc], bb[Mc][Mc];
  float acc = 0;
  for (int k = 0; k < N; k += Mc) {
    // abusing the index notation here
    // basically each thread loads one Mc x Mc element of a and b
    aa[ii][jj] = a[(i+ii)*N+k+jj];
    bb[ii][jj] = b[(k+ii)*N+j+jj];

    __syncthreads();
    for (int kk = 0; kk < Mc; kk++) {
      acc += aa[ii][kk] * bb[kk][jj];
    }
    __syncthreads();
  }

  c[(i+ii)*N+j+jj] = acc;
}


template <int N, int Mc>
__global__ void thread_blocked(float* a, float* b, float* c) {
  const int TM = Mc / 32;
  const int i = blockIdx.x * Mc;
  const int j = blockIdx.y * Mc;
  const int ii = (threadIdx.x / 32) * TM;
  const int jj = (threadIdx.x % 32) * TM;

  __shared__ float aa[Mc][Mc], bb[Mc][Mc];
  float cc[TM][TM] = {};

  for (int k = 0; k < N; k += Mc) {

    for (int iii = 0; iii < TM; iii++) {
      for (int jjj = 0; jjj < TM; jjj++) {
        aa[ii+iii][jj+jjj] = a[(i+ii+iii)*N+k+jj+jjj];
      }
    }

    for (int iii = 0; iii < TM; iii++) {
      for (int jjj = 0; jjj < TM; jjj++) {
        bb[ii+iii][jj+jjj] = b[(k+ii+iii)*N+j+jj+jjj];
      }
    }
    __syncthreads();

    for (int iii = 0; iii < TM; iii++) {
      for (int jjj = 0; jjj < TM; jjj++) {
        for (int kk = 0; kk < Mc; kk++) {
          cc[iii][jjj] += aa[ii+iii][kk] * bb[kk][jj+jjj];
        }
      }
    }
    __syncthreads();

  }

  for (int iii = 0; iii < TM; iii++) {
    for (int jjj = 0; jjj < TM; jjj++) {
      c[(i+ii+iii)*N+j+jj+jjj] = cc[iii][jjj];
    }
  }
}


template <int N, int Mc>
__global__ void thread_blocked2(float* a, float* b, float* c) {
  const int TM = Mc / 32;
  const int i = blockIdx.x * Mc;
  const int j = blockIdx.y * Mc;
  const int ii = threadIdx.x / 32;
  const int jj = threadIdx.x % 32;

  __shared__ float aa[Mc][Mc], bb[Mc][Mc];
  float cc[TM][TM] = {};

  for (int k = 0; k < N; k += Mc) {

    for (int rowb = 0; rowb < TM; rowb++) {
      for (int colb = 0; colb < TM; colb++) {
        aa[rowb*32+ii][colb*32+jj] = a[(i+(rowb*32)+ii) * N + k+(colb*32)+jj];
      }
    }

    for (int rowb = 0; rowb < TM; rowb++) {
      for (int colb = 0; colb < TM; colb++) {
        bb[rowb*32+ii][colb*32+jj] = b[(k+(rowb*32)+ii) * N + j+(colb*32)+jj];
      }
    }
    __syncthreads();

    for (int rowb = 0; rowb < TM; rowb++) {
      for (int colb = 0; colb < TM; colb++) {
        for (int kk = 0; kk < Mc; kk++) {
          cc[rowb][colb] += aa[rowb*32 + ii][kk] * bb[kk][colb*32+jj];
        }
      }
    }
    __syncthreads();

  }

  for (int rowb = 0; rowb < TM; rowb++) {
    for (int colb = 0; colb < TM; colb++) {
      c[(i+(rowb*32)+ii) * N + j + (colb * 32) + jj] = cc[rowb][colb];
    }
  }
  
}



int main() {
  const int N = 2048;

  float* a = static_cast<float*>(std::aligned_alloc(32, sizeof(float) * N * N));
  float* b = static_cast<float*>(std::aligned_alloc(32, sizeof(float) * N * N));
  float* c = static_cast<float*>(std::aligned_alloc(32, sizeof(float) * N * N));
  float* ans = static_cast<float*>(std::aligned_alloc(32, sizeof(float) * N * N));

  rand_matrix<N>(a);
  rand_matrix<N>(b);
  zero_matrix<N>(c);
  zero_matrix<N>(ans);

  baseline<N>(a, b, ans);
  //print_matrix<N>(ans);

  float* d_a;
  float* d_b;
  float* d_c;

  int matsize = N * N * sizeof(float);

  hipMalloc(&d_a, matsize);
  hipMalloc(&d_b, matsize);
  hipMalloc(&d_c, matsize);
  
  hipMemcpy(d_a, a, matsize, hipMemcpyHostToDevice);
  hipMemcpy(d_b, b, matsize, hipMemcpyHostToDevice);
  hipMemcpy(d_c, c, matsize, hipMemcpyHostToDevice);

  BENCH_KERNEL("baseline_cuda", N, baseline_cuda<N>,
               dim3(N / 32, N / 32), dim3(32, 32), d_a, d_b, d_c);
  hipMemcpy(c, d_c, matsize, hipMemcpyDeviceToHost);
  check_matrix<N>(c, ans);

  BENCH_KERNEL("gmem_coalesced", N, (gmem_coalesced<N, 32>),
               dim3(N / 32, N / 32), dim3(32 * 32), d_a, d_b, d_c);
  hipMemcpy(c, d_c, matsize, hipMemcpyDeviceToHost);
  check_matrix<N>(c, ans);

  BENCH_KERNEL("smem_blocked", N, (smem_blocked<N, 32>),
               dim3(N / 32, N / 32), dim3(32 * 32), d_a, d_b, d_c);
  hipMemcpy(c, d_c, matsize, hipMemcpyDeviceToHost);
  check_matrix<N>(c, ans);

  BENCH_KERNEL("smem_blocked2", N, (smem_blocked2<N, 32>),
               dim3(N / 32, N / 32), dim3(32 * 32), d_a, d_b, d_c);
  hipMemcpy(c, d_c, matsize, hipMemcpyDeviceToHost);
  check_matrix<N>(c, ans);


  zero_matrix<N>(c);
  BENCH_KERNEL("thread_blocked", N, (thread_blocked<N, 64>),
               dim3(N/64, N/64), dim3(1024), d_a, d_b, d_c);
  hipMemcpy(c, d_c, matsize, hipMemcpyDeviceToHost);
  check_matrix<N>(c, ans);

  zero_matrix<N>(c);
  BENCH_KERNEL("thread_blocked2", N, (thread_blocked2<N, 64>),
               dim3(N/64, N/64), dim3(1024), d_a, d_b, d_c);
  hipMemcpy(c, d_c, matsize, hipMemcpyDeviceToHost);
  //print_matrix<N>(c);
  check_matrix<N>(c, ans);



  hipFree(d_a);
  hipFree(d_b);
  hipFree(d_c);
}
