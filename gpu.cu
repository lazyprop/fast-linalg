#include "hip/hip_runtime.h"
#include <iostream>
#include <memory>
#include <chrono>

#include "util.h"
#include "matmul.h"


#define BENCH_KERNEL(name, N, kernel, grid_dim, block_dim, a, b, c)  \
  { \
    const int runs = 10; \
    double seconds = 0; \
    for (int i = 0; i < runs; i++) { \
      zero_cuda<N>(c);  \
      auto start = std::chrono::high_resolution_clock::now(); \
      kernel<<<grid_dim, block_dim>>>(a, b, c);                     \
      auto end = std::chrono::high_resolution_clock::now();            \
      std::chrono::duration<double, std::milli> ms_double = end - start; \
      seconds += ms_double.count();                                     \
    } \
    std::cout << name << ": " << (2.0 * N * N * N) / (seconds / runs / 1000 * 1e9) \
              << " GFLOPS/s\n"; \
  }



template<int N, int Mc>
__global__ void baseline_cuda_kernel(float* a, float* b, float* c) {
  int i = blockIdx.x * N + threadIdx.x;
  int j = blockIdx.y * N + threadIdx.y;
  float acc = 0;
  for (int k = 0; k < N; k++) {
    acc += a[i*N+j] * b[k*N+j];
  }
  c[i*N+j] = acc;
}

template <int N>
void baseline_cuda(float* a, float* b, float* c) {
  float* d_a;
  float* d_b;
  float* d_c;

  int matsize = N * N * sizeof(float);

  hipMalloc(&d_a, matsize);
  hipMalloc(&d_b, matsize);
  hipMalloc(&d_c, matsize);
  
  hipMemcpy(d_a, a, matsize, hipMemcpyHostToDevice);
  hipMemcpy(d_b, b, matsize, hipMemcpyHostToDevice);
  hipMemcpy(d_c, c, matsize, hipMemcpyHostToDevice);

  dim3 block_dim(32, 32, 1);
  dim3 grid_dim(N / 32, N / 32, 1);

  BENCH_KERNEL("baseline_cuda", N, (baseline_cuda_kernel<N, 32>), grid_dim, block_dim,
               d_a, d_b, d_c);

  hipMemcpy(c, d_c, matsize, hipMemcpyDeviceToHost);
}


int main() {
  const int N = 256;

  float* a = static_cast<float*>(std::aligned_alloc(32, sizeof(float) * N * N));
  float* b = static_cast<float*>(std::aligned_alloc(32, sizeof(float) * N * N));
  float* c = static_cast<float*>(std::aligned_alloc(32, sizeof(float) * N * N));
  float* ans = static_cast<float*>(std::aligned_alloc(32, sizeof(float) * N * N));

  //seq_init<N>(a);
  //seq_init<N>(b);
  rand_matrix<N>(a);
  rand_matrix<N>(b);
  zero_matrix<N>(c);
  zero_matrix<N>(ans);

  baseline<N>(a, b, ans);

  baseline_cuda<N>(a, b, c);
}