#include "hip/hip_runtime.h"
#include <iostream>
#include <memory>
#include <chrono>

#include "util.h"
#include "matmul.h"

#define CHECK_ERROR() \
  { \
    hipError_t e = hipGetLastError(); \
    std::cout << hipGetErrorName(e) << '\n'; \
    std::cout << hipGetErrorString(e) << '\n'; \
  }

#define BENCH_KERNEL(name, N, kernel, grid_dim, block_dim, d_a, d_b, d_c)  \
  { \
    const int runs = 100; \
    double ms = 0; \
    for (int i = 0; i < runs; i++) { \
      zero_cuda<N>(d_c); \
      hipDeviceSynchronize(); \
      auto start = std::chrono::high_resolution_clock::now(); \
      kernel<<<grid_dim, block_dim>>>(d_a, d_b, d_c);                     \
      hipDeviceSynchronize(); \
      auto end = std::chrono::high_resolution_clock::now();            \
      std::chrono::duration<double, std::milli> ms_double = end - start; \
      ms += ms_double.count();                                     \
    } \
    std::cout << name << ": " << (2.0 * N * N * N) / (1e9 * ms / runs / 1000) \
              << " GFLOPS/s\n"; \
  }



template<int N>
__global__ void baseline_cuda_kernel(float* a, float* b, float* c) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  int j = blockIdx.y * blockDim.y + threadIdx.y;
  float acc = 0;
  for (int k = 0; k < N; k++) {
    acc += a[i*N+k] * b[k*N+j];
  }
  c[i*N+j] = acc;
}

template <int N>
void baseline_cuda(float* a, float* b, float* c) {
  float* d_a;
  float* d_b;
  float* d_c;

  int matsize = N * N * sizeof(float);

  hipMalloc(&d_a, matsize);
  hipMalloc(&d_b, matsize);
  hipMalloc(&d_c, matsize);
  
  hipMemcpy(d_a, a, matsize, hipMemcpyHostToDevice);
  hipMemcpy(d_b, b, matsize, hipMemcpyHostToDevice);
  hipMemcpy(d_c, c, matsize, hipMemcpyHostToDevice);

  dim3 block_dim(32, 32);
  dim3 grid_dim(N / 32, N / 32);

  baseline_cuda_kernel<N><<<grid_dim, block_dim>>>(d_a, d_b, d_c);
  BENCH_KERNEL("baseline_cuda", N, baseline_cuda_kernel<N>, grid_dim, block_dim,
               d_a, d_b, d_c);

  hipMemcpy(c, d_c, matsize, hipMemcpyDeviceToHost);

  hipFree(d_a);
  hipFree(d_b);
  hipFree(d_c);
}


template<int N, int Mc>
__global__ void coalesced_kernel(float* a, float* b, float* c) {
  int i = blockIdx.x * Mc + (threadIdx.x / Mc);
  int j = blockIdx.y * Mc + (threadIdx.x % Mc);
  float acc = 0;
  for (int k = 0; k < N; k++) {
    acc += a[i*N+k] * b[k*N+j];
  }
  c[i*N+j] = acc;
}

template <int N>
void coalesced(float* a, float* b, float* c) {
  float* d_a;
  float* d_b;
  float* d_c;

  int matsize = N * N * sizeof(float);

  hipMalloc(&d_a, matsize);
  hipMalloc(&d_b, matsize);
  hipMalloc(&d_c, matsize);
  
  hipMemcpy(d_a, a, matsize, hipMemcpyHostToDevice);
  hipMemcpy(d_b, b, matsize, hipMemcpyHostToDevice);
  hipMemcpy(d_c, c, matsize, hipMemcpyHostToDevice);

  dim3 block_dim(32 * 32);
  dim3 grid_dim(N / 32, N / 32);

  BENCH_KERNEL("coalesced_cuda", N, (coalesced_kernel<N, 32>), grid_dim, block_dim,
               d_a, d_b, d_c);

  hipMemcpy(c, d_c, matsize, hipMemcpyDeviceToHost);

  hipFree(d_a);
  hipFree(d_b);
  hipFree(d_c);
}


int main() {
  const int N = 1024;

  float* a = static_cast<float*>(std::aligned_alloc(32, sizeof(float) * N * N));
  float* b = static_cast<float*>(std::aligned_alloc(32, sizeof(float) * N * N));
  float* c = static_cast<float*>(std::aligned_alloc(32, sizeof(float) * N * N));
  float* ans = static_cast<float*>(std::aligned_alloc(32, sizeof(float) * N * N));

  //seq_init<N>(a);
  //seq_init<N>(b);
  rand_matrix<N>(a);
  rand_matrix<N>(b);
  zero_matrix<N>(c);
  zero_matrix<N>(ans);

  baseline<N>(a, b, ans);
  //print_matrix<N>(ans);

  baseline_cuda<N>(a, b, c);
  check_matrix<N>(c, ans);

  zero_matrix<N>(c);
  coalesced<N>(a, b, c);
  check_matrix<N>(c, ans);
}
