#include "hip/hip_runtime.h"
#include <iostream>
#include <memory>
#include <chrono>

#include "util.h"
#include "matmul.h"

#define CHECK_ERROR() \
  { \
    hipError_t e = hipGetLastError(); \
    std::cout << hipGetErrorName(e) << '\n'; \
    std::cout << hipGetErrorString(e) << '\n'; \
  }

#define BENCH_KERNEL(name, N, kernel, grid_dim, block_dim, d_a, d_b, d_c)  \
  { \
    const int runs = 100; \
    double ms = 0; \
    for (int i = 0; i < runs; i++) { \
      zero_cuda<N>(d_c); \
      hipDeviceSynchronize(); \
      auto start = std::chrono::high_resolution_clock::now(); \
      kernel<<<grid_dim, block_dim>>>(d_a, d_b, d_c);                     \
      hipDeviceSynchronize(); \
      auto end = std::chrono::high_resolution_clock::now();            \
      std::chrono::duration<double, std::milli> ms_double = end - start; \
      ms += ms_double.count();                                     \
    } \
    std::cout << name << ": " << (2.0 * N * N * N) / (1e9 * ms / runs / 1000) \
              << " GFLOPS/s\n"; \
  }


#define DEBUG_KERNEL(name, N, kernel, grid_dim, block_dim, d_a, d_b, d_c)  \
  { \
    kernel<<<grid_dim, block_dim>>>(d_a, d_b, d_c);                     \
    hipDeviceSynchronize(); \
    CHECK_ERROR(); \
  }




template<int N>
__global__ void baseline_cuda(float* a, float* b, float* c) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  int j = blockIdx.y * blockDim.y + threadIdx.y;
  float acc = 0;
  for (int k = 0; k < N; k++) {
    acc += a[i*N+k] * b[k*N+j];
  }
  c[i*N+j] = acc;
}


template<int N, int Mc>
__global__ void gmem_coalesced(float* a, float* b, float* c) {
  int i = blockIdx.x * Mc + (threadIdx.x / Mc);
  int j = blockIdx.y * Mc + (threadIdx.x % Mc);
  float acc = 0;
  for (int k = 0; k < N; k++) {
    acc += a[i*N+k] * b[k*N+j];
  }
  c[i*N+j] = acc;
}

template<int N, int Mc>
__global__ void smem_blocked(float* a, float* b, float* c) {
  int i = blockIdx.x * Mc;
  int j = blockIdx.y * Mc;
  int ii = threadIdx.x / Mc;
  int jj = threadIdx.x % Mc;

  __shared__ float aa[Mc][Mc], bb[Mc][Mc], cc[Mc][Mc];
  cc[ii][jj] = 0;
  for (int k = 0; k < N; k += Mc) {
    // abusing the index notation here
    // basically each thread loads one Mc x Mc element of a and b
    aa[ii][jj] = a[(i+ii)*N+k+jj];
    bb[ii][jj] = b[(k+ii)*N+j+jj];

    __syncthreads();
    for (int kk = 0; kk < Mc; kk++) {
      cc[ii][jj] += aa[ii][kk] * bb[kk][jj];
    }
    __syncthreads();
  }

  c[(i+ii)*N+j+jj] = cc[ii][jj];
}

int main() {
  const int N = 1024;

  float* a = static_cast<float*>(std::aligned_alloc(32, sizeof(float) * N * N));
  float* b = static_cast<float*>(std::aligned_alloc(32, sizeof(float) * N * N));
  float* c = static_cast<float*>(std::aligned_alloc(32, sizeof(float) * N * N));
  float* ans = static_cast<float*>(std::aligned_alloc(32, sizeof(float) * N * N));

  //seq_init<N>(a);
  //seq_init<N>(b);
  rand_matrix<N>(a);
  rand_matrix<N>(b);
  zero_matrix<N>(c);
  zero_matrix<N>(ans);

  baseline<N>(a, b, ans);
  //print_matrix<N>(ans);

  float* d_a;
  float* d_b;
  float* d_c;

  int matsize = N * N * sizeof(float);

  hipMalloc(&d_a, matsize);
  hipMalloc(&d_b, matsize);
  hipMalloc(&d_c, matsize);
  
  hipMemcpy(d_a, a, matsize, hipMemcpyHostToDevice);
  hipMemcpy(d_b, b, matsize, hipMemcpyHostToDevice);
  hipMemcpy(d_c, c, matsize, hipMemcpyHostToDevice);

  BENCH_KERNEL("baseline_cuda", N, baseline_cuda<N>,
               dim3(N / 32, N / 32), dim3(32, 32), d_a, d_b, d_c);
  hipMemcpy(c, d_c, matsize, hipMemcpyDeviceToHost);
  check_matrix<N>(c, ans);

  BENCH_KERNEL("gmem_coalesced", N, (gmem_coalesced<N, 32>),
               dim3(N / 32, N / 32), dim3(32 * 32), d_a, d_b, d_c);
  hipMemcpy(c, d_c, matsize, hipMemcpyDeviceToHost);
  check_matrix<N>(c, ans);

  BENCH_KERNEL("smem_blocked", N, (smem_blocked<N, 32>),
               dim3(N / 32, N / 32), dim3(32 * 32), d_a, d_b, d_c);
  hipMemcpy(c, d_c, matsize, hipMemcpyDeviceToHost);
  check_matrix<N>(c, ans);


  hipFree(d_a);
  hipFree(d_b);
  hipFree(d_c);
}
